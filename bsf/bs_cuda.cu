#include <stdio.h>
#include <inttypes.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__device__ double c[5] = {0.31938153, -0.356563782, 1.781477937, -1.821255978, 1.33027442};
__device__ double r = 0.02;
__device__ double v = 0.03;

#define CNDP2m(R,D)\
{\
	double K, L;\
	\
	K = 1 / (1 + 0.2316419 * (L = fabs(D)));\
	R = 0;\
	R += c[0] * pow(K, 1);\
	R += c[1] * pow(K, 2);\
	R += c[2] * pow(K, 3);\
	R += c[3] * pow(K, 4);\
	R += c[4] * pow(K, 5);\
	R = 0.3989422804 * exp((L * L) / -2) * R;\
	R = ((D >= 0) ? -1 * (-1 + R) : R);\
}

__global__ void
kernel(double *res, int64_t *S, int64_t *X, double *T, size_t count)
{
	int i;
	double expRT, vsqrtT, D1, D2, CD1, CD2;
	
	i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < count) {	
		expRT = exp(-r * T[i]);
		vsqrtT = v*pow(T[i], 0.5);
		D1 = log(((double)S[i]) / ((double)X[i]));
		D1 = (D1 + (r + (v*v) / 2) * T[i]) / vsqrtT;
		D2 = D1 - vsqrtT;
		CNDP2m(CD2,D2);
		CNDP2m(CD1,D1);
		res[i*2] = (S[i] * CD1) - X[i] * expRT * CD2;
		res[i*2+1] = (X[i] * expRT * (1 - CD2)) - S[i] * (1 - CD1);
	}
}

extern "C" void
copy_data(double **res, int64_t **Sp, int64_t **Xp, double **Tp, int64_t *S, int64_t *X, int64_t *T, size_t count)
{
	*Tp = (double*)malloc(count * sizeof(double));
	*Sp = (int64_t*)malloc(count * sizeof(int64_t));
	*Xp = (int64_t*)malloc(count * sizeof(int64_t));
	*res = (double*)malloc (2 * count * sizeof(double));
	
	memcpy(*Tp,T,count*sizeof(double));
	memcpy(*Sp,S,count*sizeof(int64_t));
	memcpy(*Xp,X,count*sizeof(int64_t));
}

#define chk(m,x) if(hipSuccess != (x)){printf("\n\n\n\n\n\n\nFailure: %s!\n",(m));return 1;}

extern "C" int64_t
bs_c(double *res, int64_t *S, int64_t *X, double *T, size_t count)
{
	int bs,tc;
	size_t db,ib;
	double *gres, *gT;
	int64_t *gS, *gX;
	
	ib=count*sizeof(int64_t);db=count*sizeof(double);
	
	chk("gres alloc",hipMalloc(&gres,2*db));
	chk("gS alloc",hipMalloc(&gS,ib));
	chk("gX alloc",hipMalloc(&gX,ib));
	chk("gT alloc",hipMalloc(&gT,db));
	
	chk("gS copy",hipMemcpy(gS,S,ib,hipMemcpyHostToDevice));
	chk("gX copy",hipMemcpy(gX,X,ib,hipMemcpyHostToDevice));
	chk("gT copy",hipMemcpy(gT,T,db,hipMemcpyHostToDevice));
	
	tc = 896;
	bs = (count+tc-1)/tc;
	kernel<<<bs,tc>>>(gres,gS,gX,gT,count);
	
	chk("res copy",hipMemcpy(res,gres,2*db,hipMemcpyDeviceToHost));

	hipFree(gres);hipFree(gS);hipFree(gX);hipFree(gT);
	
	return 0;
}

