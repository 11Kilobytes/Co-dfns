#include "hip/hip_runtime.h"
#include "h.cuh"

__device__ type_d coe[]={0.31938153,-0.356563782,1.781477937,-1.821255978,1.33027442};

__global__ V redk(D*zp,D*wp,UI64 sz){UI64 i=blockDim.x*blockIdx.x+threadIdx.x;
 if(i<sz){D w=wp[i];D z=coe[0]*w*1;z+=coe[1]*w*2;z+=coe[2]*w*3;
  z+=coe[3]*w*4;z+=coe[4]*w*5;zp[i]=z;}}

extern "C" {
UDF(codfns_coeffred){h2g(rgt);D*ze;prep((V**)&ze,res,rgt);
 UI64 bs=(siz(res)+1024-1)/1024;ze=(D*)gpu(res);D*re=(D*)gpu(rgt);
 redk<<<bs,1024>>>(ze,re,siz(res));typ(res)=apl_type_d;ong(res)=1;R 0;}
}
