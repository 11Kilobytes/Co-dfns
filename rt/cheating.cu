#include "hip/hip_runtime.h"
#include "h.cuh"

__device__ type_d coeff[]={0.31938153,-0.356563782,1.781477937,-1.821255978,1.33027442};

__global__ V redk(D*zp,D*wp,UI64 sz){UI64 i=blockDim.x*blockIdx.x+threadIdx.x;
 if(i<sz){D w=wp[i];D z=coeff[0]*w*1;z+=coeff[1]*w*2;z+=coeff[2]*w*3;
  z+=coeff[3]*w*4;z+=coeff[4]*w*5;zp[i]=z;}}

extern "C" {
UDF(codfns_coeffred){h2g(rgt);D*ze;prep((V**)&ze,res,rgt);
 UI64 bs=(siz(res)+1024-1)/1024;ze=(D*)gpu(res);D*re=(D*)gpu(rgt);
 redk<<<bs,1024>>>(ze,re,siz(res));typ(res)=apl_type_d;ong(res)=1;R 0;}
}
