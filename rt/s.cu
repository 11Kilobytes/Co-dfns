#include "hip/hip_runtime.h"
#include "h.cuh"

extern "C" {
/* SCALAR PRIMITIVES */
/*  +A */ scalar_monadic(addm,i,d,z[i]=r[i]) 
/* A+B */ scalar_dyadic(addd,d,d,d,i,{z[i]=l[i%sl]+r[i%sr];}) 
/*  ÷A */ scalar_monadic(dividem,d,d,{
 if(r[i]!=0){z[i]=1.0/r[i];}else ERR(11,"DOMAIN ERROR: Divide by zero\n");})
/* A÷B */ scalar_dyadic(divided,d,d,d,d,{
 if(r[i%sr]!=0){z[i]=(1.0*l[i%sl])/(1.0*r[i%sr]);}
 else ERR(11,"DOMAIN ERROR: Divide by zero\n");})
/* A=B */ scalar_dyadic(equald,i,i,i,i,{z[i]=(l[i%sl]==r[i%sr]);})
/* A≥B */ scalar_dyadic(greateqd,i,i,i,i,{z[i]=(l[i%sl]>=r[i%sr]);})
/* A>B */ scalar_dyadic(greaterd,i,i,i,i,{z[i]=(l[i%sl]>r[i%sr]);})
/* A≤B */ scalar_dyadic(lesseqd,i,i,i,i,{z[i]=(l[i%sl]<=r[i%sr]);})
/* A<B */ scalar_dyadic(lessd,i,i,i,i,{z[i]=(l[i%sl]<r[i%sr]);})
/*  ⍟A */ scalar_monadic(logm,d,d,z[i]=log((D)r[i]))
/* A⍟B */ scalar_dyadic(logd,d,d,d,d,{z[i]=log((D)r[i%sr])/log((D)l[i%sl]);})
/*  |A */ smi(residuem,i,i,z[i]=labs(r[i]))
/*  |A */ smi(residuem,d,d,z[i]=fabs(r[i]))
/*  |A */ scalar_monadic_main(residuem,d,i)
#define RESIDUED {z[i]=fmod(r[i%sr],l[i%sl]);}
/* A|B */ sdi(residued,d,d,d,RESIDUED)
/* A|B */ sdi(residued,d,d,i,RESIDUED)
/* A|B */ sdi(residued,d,i,d,RESIDUED)
/* A|B */ sdi(residued,i,i,i,z[i]=l[i%sl]%r[i%sr];)
/* A|B */ scalar_dyadic_main(residued,d,d,d,i)
/*  ⌈B */ scalar_monadic(maxm,d,i,z[i]=ceil((D)r[i]))
/* A⌈B */ scalar_dyadic(maxd,d,d,d,i,{z[i]=(l[i%sl]>=r[i%sr]?l[i%sl]:r[i%sr]);})
/*  ⌊A */ scalar_monadic(minm,d,i,z[i]=floor((D)r[i]))
/* A⌊B */ scalar_dyadic(mind,d,d,d,i,{z[i]=(l[i%sl]<=r[i%sr]?l[i%sl]:r[i%sr]);})
/*  ×A */ scalar_monadic(multiplym,d,i,{
 if(r[i]==0)z[i]=0;else if(r[i]<0)z[i]=-1;else z[i]=1;})
/* A×B */ scalar_dyadic(multiplyd,d,d,d,i,{z[i]=l[i%sl] * r[i%sr];})
/* A≠B */ scalar_dyadic(neqd,i,i,i,i,{z[i]=(l[i%sl]!=r[i%sr]);})
/*  ~A */ scalar_monadic(notm,i,i,{
 if(r[i]==1){z[i]=0;}else if(r[i]==0){z[i]=1;}else{ERR(11,"DOMAIN ERROR\n");}})
/*  *A */ scalar_monadic(powerm,d,i,z[i]=exp((D)r[i]))
/* A*B */ scalar_dyadic(powerd,d,d,d,i,{z[i]=pow(l[i%sl],r[i%sr]);})
/*  -A */ scalar_monadic(subtractm,d,i,z[i]=-1 * r[i])
/* A-B */ scalar_dyadic(subtractd,d,d,d,i,{z[i]=l[i%sl]-r[i%sr];})
}
